#include "hip/hip_runtime.h"
#include "dpd-forces.h"
#include "common.h"

__device__ bool inbox(float x, float y, float z,
        float xl=-3, float xh=3,
        float yl=-3, float yh=3,
        float zl=-3, float zh=3) {
    return xl < x && x < xh && yl < y && y < yh  && zl < z && z < zh;
}

__device__ float3 compute_dpd_force_traced(int type1, int type2,
        float3 pos1, float3 pos2, float3 vel1, float3 vel2, float myrandnr) {
    /* return the DPD interaction force based on particle types
     * type: 0 -- outer solvent, 1 -- inner solvent, 2 -- membrane, 3 -- wall */

    /************** User-defined constants ***********/
    // All particles: DPD interactions
    const float gammadpd[4] = {8., 8., 8., 8.};  // default: 8
    const float aij[4] = {4. / RC_FX, 4. / RC_FX, 4. / RC_FX, 4. / RC_FX};  // default: 4
    const float gammadpd_pair = 0.5 * (gammadpd[type1] + gammadpd[type2]);
    const float sigmaf_pair = sqrt(2*gammadpd_pair*kBT / dt);

    // RBC-RBC: Lennard-Jones interactions
    const float ljsigma = 0.3;  // default: 0.3
    const float ljepsilon = 1.0 / (RC_FX*RC_FX);  // default: 1
    /********** End of User-defined constants ********/


    const float _xr = pos1.x - pos2.x;
    const float _yr = pos1.y - pos2.y;
    const float _zr = pos1.z - pos2.z;

    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
    const float invrij = rsqrtf(rij2);
    const float rij = rij2 * invrij;
    if (rij2 >= RC_FX)
        return make_float3(0, 0, 0);

    const float argwr = 1.f - rij;
    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

    const float xr = _xr * invrij;
    const float yr = _yr * invrij;
    const float zr = _zr * invrij;

    const float rdotv =
        xr * (vel1.x - vel2.x) +
        yr * (vel1.y - vel2.y) +
        zr * (vel1.z - vel2.z);

    float strength = (-gammadpd_pair * wr * rdotv + sigmaf_pair * myrandnr) * wr;
    if (type1 == 2 && type2 == 2) {  // membrane contact
        const float invr2 = invrij * invrij;
        const float t2 = ljsigma * ljsigma * invr2;
        const float t4 = t2 * t2;
        const float t6 = t4 * t2;
        const float lj = min(1e4f, max(0.f, ljepsilon * 24.f * invrij * t6 * (2.f * t6 - 1.f)));
        strength += lj;
    } else {
        const float aij_pair = 0.5 * (aij[type1] + aij[type2]);
        strength += aij_pair * argwr;
    }

    return make_float3(strength*xr, strength*yr, strength*zr);
}
