#include "hip/hip_runtime.h"
#include <dpd-rng.h>
#include <vector>
#include <cstdio>
#include <mpi.h>
#include ".conf.h" /* configuration file (copy from .conf.test.h) */
#include "common.h"
#include "common-kernels.h"
#include "fsi.h"
#include "dpd-forces.h"
#include "last_bit_float.h"

ComputeFSI::ComputeFSI(MPI_Comm comm)
{
    int myrank;
    MPI_CHECK( MPI_Comm_rank(comm, &myrank));

    local_trunk = Logistic::KISS(1908 - myrank, 1409 + myrank, 290, 12968);

    CC(hipPeekAtLastError());
}

namespace KernelsFSI
{
    texture<float2, hipTextureType1D> texSolventParticles;
    texture<int, hipTextureType1D> texCellsStart, texCellsCount;

    bool firsttime = true;

    static const int NCELLS = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

    __global__
        void interactions_3tpp(const float2 * const particles, const int np, const int nsolvent,
                float * const acc, float * const accsolvent, const float seed)
        {
#if !defined(__CUDA_ARCH__)
#warning __CUDA_ARCH__ not defined! assuming 350
#define _ACCESS(x) __ldg(x)
#elif __CUDA_ARCH__ >= 350
#define _ACCESS(x) __ldg(x)
#else
#define _ACCESS(x) (*(x))
#endif

            const int gid = threadIdx.x + blockDim.x * blockIdx.x;
            const int pid = gid / 3;
            const int zplane = gid % 3;

            if (pid >= np)
                return;

            const float2 dst0 = _ACCESS(particles + 3 * pid + 0);
            const float2 dst1 = _ACCESS(particles + 3 * pid + 1);
            const float2 dst2 = _ACCESS(particles + 3 * pid + 2);

            int scan1, scan2, ncandidates, spidbase;
            int deltaspid1, deltaspid2;

            {
                enum
                {
                    XCELLS = XSIZE_SUBDOMAIN,
                    YCELLS = YSIZE_SUBDOMAIN,
                    ZCELLS = ZSIZE_SUBDOMAIN,
                    XOFFSET = XCELLS / 2,
                    YOFFSET = YCELLS / 2,
                    ZOFFSET = ZCELLS / 2
                };

                const int xcenter = XOFFSET + (int)floorf(dst0.x);
                const int xstart = max(0, xcenter - 1);
                const int xcount = min(XCELLS, xcenter + 2) - xstart;

                if (xcenter - 1 >= XCELLS || xcenter + 2 <= 0)
                    return;

                const int ycenter = YOFFSET + (int)floorf(dst0.y);

                const int zcenter = ZOFFSET + (int)floorf(dst1.x);
                const int zmy = zcenter - 1 + zplane;
                const bool zvalid = zmy >= 0 && zmy < ZCELLS;

                int count0 = 0, count1 = 0, count2 = 0;

                if (zvalid && ycenter - 1 >= 0 && ycenter - 1 < YCELLS)
                {
                    const int cid0 = xstart + XCELLS * (ycenter - 1 + YCELLS * zmy);
                    spidbase = tex1Dfetch(texCellsStart, cid0);
                    count0 = ((cid0 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid0 + xcount)) - spidbase;
                }

                if (zvalid && ycenter >= 0 && ycenter < YCELLS)
                {
                    const int cid1 = xstart + XCELLS * (ycenter + YCELLS * zmy);
                    deltaspid1 = tex1Dfetch(texCellsStart, cid1);
                    count1 = ((cid1 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid1 + xcount)) - deltaspid1;
                }

                if (zvalid && ycenter + 1 >= 0 && ycenter + 1 < YCELLS)
                {
                    const int cid2 = xstart + XCELLS * (ycenter + 1 + YCELLS * zmy);
                    deltaspid2 = tex1Dfetch(texCellsStart, cid2);
                    count2 = ((cid2 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid2 + xcount)) - deltaspid2;
                }

                scan1 = count0;
                scan2 = count0 + count1;
                ncandidates = scan2 + count2;

                deltaspid1 -= scan1;
                deltaspid2 -= scan2;
            }

            float xforce = 0, yforce = 0, zforce = 0;

#pragma unroll 3
            for(int i = 0; i < ncandidates; ++i)
            {
                const int m1 = (int)(i >= scan1);
                const int m2 = (int)(i >= scan2);
                const int spid = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);

                const int sentry = 3 * spid;
                const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry    );
                const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
                const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);

                const float myrandnr = Logistic::mean0var1(seed, pid, spid);

                // check for particle types and compute the DPD force
                float3 pos1 = make_float3(dst0.x, dst0.y, dst1.x), pos2 = make_float3(stmp0.x, stmp0.y, stmp1.x);
                float3 vel1 = make_float3(dst1.y, dst2.x, dst2.y), vel2 = make_float3(stmp1.y, stmp2.x, stmp2.y);
                int type1 = MEMB_TYPE;  // RBC membrane
                int type2 = last_bit_float::get(vel2.x) ? IN_TYPE : OUT_TYPE;
                const float3 strength = compute_dpd_force_traced(type1, type2,
                        pos1, pos2, vel1, vel2, myrandnr);

                const float xinteraction = strength.x;
                const float yinteraction = strength.y;
                const float zinteraction = strength.z;

                xforce += xinteraction;
                yforce += yinteraction;
                zforce += zinteraction;

                atomicAdd(accsolvent + sentry    , -xinteraction);
                atomicAdd(accsolvent + sentry + 1, -yinteraction);
                atomicAdd(accsolvent + sentry + 2, -zinteraction);
            }

            atomicAdd(acc + 3 * pid + 0, xforce);
            atomicAdd(acc + 3 * pid + 1, yforce);
            atomicAdd(acc + 3 * pid + 2, zforce);
        }

    void setup(const Particle * const solvent, const int npsolvent, const int * const cellsstart, const int * const cellscount)
    {
        if (firsttime)
        {
            texCellsStart.channelDesc = hipCreateChannelDesc<int>();
            texCellsStart.filterMode = hipFilterModePoint;
            texCellsStart.mipmapFilterMode = hipFilterModePoint;
            texCellsStart.normalized = 0;

            texCellsCount.channelDesc = hipCreateChannelDesc<int>();
            texCellsCount.filterMode = hipFilterModePoint;
            texCellsCount.mipmapFilterMode = hipFilterModePoint;
            texCellsCount.normalized = 0;

            texSolventParticles.channelDesc = hipCreateChannelDesc<float2>();
            texSolventParticles.filterMode = hipFilterModePoint;
            texSolventParticles.mipmapFilterMode = hipFilterModePoint;
            texSolventParticles.normalized = 0;

            CC(hipFuncSetCacheConfig(reinterpret_cast<const void*>(interactions_3tpp), hipFuncCachePreferL1));

            firsttime = false;
        }

        size_t textureoffset = 0;

        if (npsolvent)
        {
            CC(hipBindTexture(&textureoffset, &texSolventParticles, solvent, &texSolventParticles.channelDesc,
                        sizeof(float) * 6 * npsolvent));
        }

        const int ncells = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;

        CC(hipBindTexture(&textureoffset, &texCellsStart, cellsstart, &texCellsStart.channelDesc, sizeof(int) * ncells));

        CC(hipBindTexture(&textureoffset, &texCellsCount, cellscount, &texCellsCount.channelDesc, sizeof(int) * ncells));
    }
}

void ComputeFSI::bulk(std::vector<ParticlesWrap> wsolutes, hipStream_t stream)
{
    if (wsolutes.size() == 0)
        return;

    KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart, wsolvent.cellscount);

    CC(hipPeekAtLastError());

    for(std::vector<ParticlesWrap>::iterator it = wsolutes.begin(); it != wsolutes.end(); ++it)
        if (it->n)
            KernelsFSI::interactions_3tpp<<< (3 * it->n + 127) / 128, 128, 0, stream >>>
                ((float2 *)it->p, it->n, wsolvent.n, (float *)it->a, (float *)wsolvent.a, local_trunk.get_float());

    CC(hipPeekAtLastError());
}

namespace KernelsFSI
{
    __constant__ int packstarts_padded[27], packcount[26];
    __constant__ Particle * packstates[26];
    __constant__ Acceleration * packresults[26];

    __global__ void interactions_halo(const int nparticles_padded, const int nsolvent, float * const accsolvent, const float seed)
    {
        const int laneid = threadIdx.x & 0x1f;
        const int warpid = threadIdx.x >> 5;
        const int localbase = 32 * (warpid + 4 * blockIdx.x);
        const int pid = localbase + laneid;

        if (localbase >= nparticles_padded)
            return;

        int nunpack;
        float2 dst0, dst1, dst2;
        float * dst = NULL;

        {
            const uint key9 = 9 * (localbase >= packstarts_padded[9]) + 9 * (localbase >= packstarts_padded[18]);
            const uint key3 = 3 * (localbase >= packstarts_padded[key9 + 3]) + 3 * (localbase >= packstarts_padded[key9 + 6]);
            const uint key1 = (localbase >= packstarts_padded[key9 + key3 + 1]) + (localbase >= packstarts_padded[key9 + key3 + 2]);
            const int code = key9 + key3 + key1;
            const int unpackbase = localbase - packstarts_padded[code];

            nunpack = min(32, packcount[code] - unpackbase);

            if (nunpack == 0)
                return;

            read_AOS6f((float2 *)(packstates[code] + unpackbase), nunpack, dst0, dst1, dst2);

            dst = (float*)(packresults[code] + unpackbase);
        }

        float xforce = 0, yforce = 0, zforce = 0;

        const int nzplanes = laneid < nunpack ? 3 : 0;

        for(int zplane = 0; zplane < nzplanes; ++zplane)
        {
            int scan1, scan2, ncandidates, spidbase;
            int deltaspid1, deltaspid2;

            {
                enum
                {
                    XCELLS = XSIZE_SUBDOMAIN,
                    YCELLS = YSIZE_SUBDOMAIN,
                    ZCELLS = ZSIZE_SUBDOMAIN,
                    XOFFSET = XCELLS / 2,
                    YOFFSET = YCELLS / 2,
                    ZOFFSET = ZCELLS / 2
                };

                const int NCELLS = XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN;
                const int xcenter = XOFFSET + (int)floorf(dst0.x);
                const int xstart = max(0, xcenter - 1);
                const int xcount = min(XCELLS, xcenter + 2) - xstart;

                if (xcenter - 1 >= XCELLS || xcenter + 2 <= 0)
                    continue;

                const int ycenter = YOFFSET + (int)floorf(dst0.y);

                const int zcenter = ZOFFSET + (int)floorf(dst1.x);
                const int zmy = zcenter - 1 + zplane;
                const bool zvalid = zmy >= 0 && zmy < ZCELLS;

                int count0 = 0, count1 = 0, count2 = 0;

                if (zvalid && ycenter - 1 >= 0 && ycenter - 1 < YCELLS)
                {
                    const int cid0 = xstart + XCELLS * (ycenter - 1 + YCELLS * zmy);
                    spidbase = tex1Dfetch(texCellsStart, cid0);
                    count0 = ((cid0 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid0 + xcount)) - spidbase;
                }

                if (zvalid && ycenter >= 0 && ycenter < YCELLS)
                {
                    const int cid1 = xstart + XCELLS * (ycenter + YCELLS * zmy);
                    deltaspid1 = tex1Dfetch(texCellsStart, cid1);
                    count1 = ((cid1 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid1 + xcount)) - deltaspid1;
                }

                if (zvalid && ycenter + 1 >= 0 && ycenter + 1 < YCELLS)
                {
                    const int cid2 = xstart + XCELLS * (ycenter + 1 + YCELLS * zmy);
                    deltaspid2 = tex1Dfetch(texCellsStart, cid2);
                    count2 = ((cid2 + xcount == NCELLS) ? nsolvent : tex1Dfetch(texCellsStart, cid2 + xcount)) - deltaspid2;
                }

                scan1 = count0;
                scan2 = count0 + count1;
                ncandidates = scan2 + count2;

                deltaspid1 -= scan1;
                deltaspid2 -= scan2;
            }

            for(int i = 0; i < ncandidates; ++i)
            {
                const int m1 = (int)(i >= scan1);
                const int m2 = (int)(i >= scan2);
                const int spid = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);

                const int sentry = 3 * spid;
                const float2 stmp0 = tex1Dfetch(texSolventParticles, sentry    );
                const float2 stmp1 = tex1Dfetch(texSolventParticles, sentry + 1);
                const float2 stmp2 = tex1Dfetch(texSolventParticles, sentry + 2);

                const float myrandnr = Logistic::mean0var1(seed, pid, spid);

                // check for particle types and compute the DPD force
                float3 pos1 = make_float3(dst0.x, dst0.y, dst1.x), pos2 = make_float3(stmp0.x, stmp0.y, stmp1.x);
                float3 vel1 = make_float3(dst1.y, dst2.x, dst2.y), vel2 = make_float3(stmp1.y, stmp2.x, stmp2.y);
                int type1 = MEMB_TYPE;  // RBC membrane
                int type2 = last_bit_float::get(vel2.x) ? IN_TYPE : OUT_TYPE;
                const float3 strength = compute_dpd_force_traced(type1, type2,
                        pos1, pos2, vel1, vel2, myrandnr);

                const float xinteraction = strength.x;
                const float yinteraction = strength.y;
                const float zinteraction = strength.z;

                xforce += xinteraction;
                yforce += yinteraction;
                zforce += zinteraction;

                atomicAdd(accsolvent + sentry    , -xinteraction);
                atomicAdd(accsolvent + sentry + 1, -yinteraction);
                atomicAdd(accsolvent + sentry + 2, -zinteraction);
            }
        }

        write_AOS3f(dst, nunpack, xforce, yforce, zforce);
    }
}

void ComputeFSI::halo(ParticlesWrap halos[26], hipStream_t stream)
{
    KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart, wsolvent.cellscount);

    CC(hipPeekAtLastError());

    int nremote_padded = 0;

    {
        int recvpackcount[26], recvpackstarts_padded[27];

        for(int i = 0; i < 26; ++i)
            recvpackcount[i] = halos[i].n;

        CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packcount), recvpackcount,
                    sizeof(recvpackcount), 0, hipMemcpyHostToDevice, stream));

        recvpackstarts_padded[0] = 0;
        for(int i = 0, s = 0; i < 26; ++i)
            recvpackstarts_padded[i + 1] = (s += 32 * ((halos[i].n + 31) / 32));

        nremote_padded = recvpackstarts_padded[26];

        CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packstarts_padded), recvpackstarts_padded,
                    sizeof(recvpackstarts_padded), 0, hipMemcpyHostToDevice, stream));
    }

    {
        const Particle * recvpackstates[26];

        for(int i = 0; i < 26; ++i)
            recvpackstates[i] = halos[i].p;

        CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packstates), recvpackstates,
                    sizeof(recvpackstates), 0, hipMemcpyHostToDevice, stream));
    }

    {
        Acceleration * packresults[26];

        for(int i = 0; i < 26; ++i)
            packresults[i] = halos[i].a;

        CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packresults), packresults,
                    sizeof(packresults), 0, hipMemcpyHostToDevice, stream));
    }

    if(nremote_padded)
        KernelsFSI::interactions_halo<<< (nremote_padded + 127) / 128, 128, 0, stream>>>
            (nremote_padded, wsolvent.n, (float *)wsolvent.a, local_trunk.get_float());

    CC(hipPeekAtLastError());
}
