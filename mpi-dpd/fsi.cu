#include "hip/hip_runtime.h"
#include <dpd-rng.h>
#include <vector>
#include <cstdio>
#include <mpi.h>
#include ".conf.h" /* configuration file (copy from .conf.test.h) */
#include "common.h"
#include "common-kernels.h"
#include "fsi.h"
#include "dpd-forces.h"
#include "last_bit_float.h"

#include "kernelsfsi.decl.h"
#include "kernelsfsi.impl.h"

ComputeFSI::ComputeFSI(MPI_Comm comm) {
  int myrank;
  MC(MPI_Comm_rank(comm, &myrank));
  local_trunk = Logistic::KISS(1908 - myrank, 1409 + myrank, 290, 12968);
  CC(hipPeekAtLastError());
}


void ComputeFSI::bulk(std::vector<ParticlesWrap> wsolutes,
                      hipStream_t stream) {
  if (wsolutes.size() == 0) return;

  KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart,
                    wsolvent.cellscount);

  CC(hipPeekAtLastError());

  for (std::vector<ParticlesWrap>::iterator it = wsolutes.begin();
       it != wsolutes.end(); ++it)
    if (it->n)
      KernelsFSI::
	interactions_3tpp<<<(3 * it->n + 127) / 128, 128, 0, stream>>>
	((float2 *)it->p, it->n, wsolvent.n, (float *)it->a,
	 (float *)wsolvent.a, local_trunk.get_float());

  CC(hipPeekAtLastError());
}

void ComputeFSI::halo(ParticlesWrap halos[26], hipStream_t stream) {
  KernelsFSI::setup(wsolvent.p, wsolvent.n, wsolvent.cellsstart,
                    wsolvent.cellscount);

  CC(hipPeekAtLastError());

  int nremote_padded = 0;

  {
    int recvpackcount[26], recvpackstarts_padded[27];

    for (int i = 0; i < 26; ++i) recvpackcount[i] = halos[i].n;

    CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packcount), recvpackcount,
                               sizeof(recvpackcount), 0, hipMemcpyHostToDevice,
                               stream));

    recvpackstarts_padded[0] = 0;
    for (int i = 0, s = 0; i < 26; ++i)
      recvpackstarts_padded[i + 1] = (s += 32 * ((halos[i].n + 31) / 32));

    nremote_padded = recvpackstarts_padded[26];

    CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(
			       KernelsFSI::packstarts_padded), recvpackstarts_padded,
			       sizeof(recvpackstarts_padded), 0, hipMemcpyHostToDevice, stream));
  }

  {
    const Particle *recvpackstates[26];

    for (int i = 0; i < 26; ++i) recvpackstates[i] = halos[i].p;

    CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packstates), recvpackstates,
                               sizeof(recvpackstates), 0,
                               hipMemcpyHostToDevice, stream));
  }

  {
    Acceleration *packresults[26];

    for (int i = 0; i < 26; ++i) packresults[i] = halos[i].a;

    CC(hipMemcpyToSymbolAsync(HIP_SYMBOL(KernelsFSI::packresults), packresults,
                               sizeof(packresults), 0, hipMemcpyHostToDevice,
                               stream));
  }

  if (nremote_padded)
    KernelsFSI::
      interactions_halo<<<(nremote_padded + 127) / 128, 128, 0, stream>>>
      (nremote_padded, wsolvent.n, (float *)wsolvent.a,
       local_trunk.get_float());

  CC(hipPeekAtLastError());
}
