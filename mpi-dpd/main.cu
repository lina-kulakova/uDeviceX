#include <cstdio>
#include <csignal>
#include <mpi.h>
#include <errno.h>
#include <dpd-rng.h>
#include <map>
#include ".conf.h" /* configuration file (copy from .conf.test.h) */
#include "m.h"     /* MPI */
#include "common.h"
#include "common.tmp.h"
#include "bund.h"

int main(int argc, char **argv) {
  m::dims[0] = m::dims[1] = m::dims[2] = 1;
  for (int iarg = 1; iarg < argc && iarg <= 3; iarg++)
    m::dims[iarg - 1] = atoi(argv[iarg]);

  int device = 0;
  CC(hipSetDevice(device));
  
  MC(MPI_Init(&argc, &argv));
  MC(MPI_Comm_rank(MPI_COMM_WORLD, &m::rank));
  MC(MPI_Cart_create(MPI_COMM_WORLD, m::d, m::dims, m::periods, 0, &m::cart));
  MC(MPI_Cart_coords(m::cart, m::rank, m::d,    m::coords));

  sim::init();
  sim::run();
  sim::close();
  
  MC(MPI_Comm_free(&m::cart));
  MC(MPI_Finalize());
}
