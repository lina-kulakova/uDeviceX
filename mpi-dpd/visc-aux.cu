#include "hip/hip_runtime.h"
/* helper functions for DPD MSD calculations */
#include "visc-aux.h"
#include "common.h"
#include "last_bit_float.h"

extern float RBCscale;

bool is_inside_rbc(float x, float y, float z, float th) {
    x *= th; y *= th; z *= th;

    float a0 = 0.0518, a1 = 2.0026, a2 = -4.491;
    float D0 = 7.82;

    float rho = (x*x+y*y)/(D0*D0);
    float s = 1-4*rho;
    if (s < 0)
        return false;

    float zrbc = D0*sqrt(s)*(a0 + a1*rho + a2*pow(rho,2));

    return z > -zrbc && z < zrbc;
}

void set_traced_particles(int n, Particle * particles) {
    for (int i = 0; i<n; i++)
        last_bit_float::set(particles[i].u[0], false);

    for (int i = 0; i<n; i++) {
        float x = particles[i].x[0];
        float y = particles[i].x[1];
        float z = particles[i].x[2];
        if (is_inside_rbc(x, y, z, 1.1/RBCscale))
            last_bit_float::set(particles[i].u[0], true);
    }
}

std::vector<int> get_traced_list(int n, Particle * const particles) {
    std::vector<int> ilist;
    for (int i = 0; i<n; i++) {
        const bool traced = last_bit_float::get(particles[i].u[0]);
        if (traced)
            ilist.push_back(i);
    }
    return ilist;
}

void print_traced_particles(Particle * particles, int n) {
    int count = 0;
    for(int i = 0; i < n; ++i) {
        bool traced = last_bit_float::get(particles[i].u[0]);
        if (traced) count++;
    }
    printf("%d particles are traced\n", count);
}

__device__ float3 compute_dpd_force_traced(int type1, int type2,
        float3 pos1, float3 pos2, float3 vel1, float3 vel2, float myrandnr) {
    /* return the DPD interaction force based on particle types
type: 0 -- outer solvent, 1 -- inner solvent, 2 -- membrane, 3 -- wall */

    const float _xr = pos1.x - pos2.x;
    const float _yr = pos1.y - pos2.y;
    const float _zr = pos1.z - pos2.z;

    const float rij2 = _xr * _xr + _yr * _yr + _zr * _zr;
    const float invrij = rsqrtf(rij2);
    const float rij = rij2 * invrij;
    if (rij2 >= 1)
        return make_float3(0, 0, 0);

    const float argwr = 1.f - rij;
    const float wr = viscosity_function<-VISCOSITY_S_LEVEL>(argwr);

    const float xr = _xr * invrij;
    const float yr = _yr * invrij;
    const float zr = _zr * invrij;

    const float rdotv =
        xr * (vel1.x - vel2.x) +
        yr * (vel1.y - vel2.y) +
        zr * (vel1.z - vel2.z);

    // particle type dependent constants
    const float gammadpd[4] = {56, 8, 56, 56};              // default: 4.5
    const float aij[4] = {4 / RC_FX, 4 / RC_FX, 4 / RC_FX, 4 / RC_FX}; // default: 75*kBT/numberdensity -- Groot and Warren (1997)

    const float aij_pair = 0.5 * (aij[type1] + aij[type2]);
    const float gammadpd_pair = 0.5 * (gammadpd[type1] + gammadpd[type2]);
    const float sigmaf_pair = sqrt(2*gammadpd_pair*kBT / dt);

    const float strength = aij_pair * argwr + (-gammadpd_pair * wr * rdotv + sigmaf_pair * myrandnr) * wr;

    return make_float3(strength*xr, strength*yr, strength*zr);
}
