#include "hip/hip_runtime.h"
/*
 *  wall.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2014-11-19.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <sys/stat.h>
#include <sys/types.h>

#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <mpi.h>
#include "common.h"
#include "io.h"
#include <dpd-rng.h>
#include "wall.h"
#include "redistancing.h"
#include "dpd-forces.h"
#include "last_bit_float.h"

enum {
  XSIZE_WALLCELLS = 2 * XMARGIN_WALL + XSIZE_SUBDOMAIN,
  YSIZE_WALLCELLS = 2 * YMARGIN_WALL + YSIZE_SUBDOMAIN,
  ZSIZE_WALLCELLS = 2 * ZMARGIN_WALL + ZSIZE_SUBDOMAIN,

  XTEXTURESIZE = 256,

  _YTEXTURESIZE = ((YSIZE_SUBDOMAIN + 2 * YMARGIN_WALL) * XTEXTURESIZE +
		   XSIZE_SUBDOMAIN + 2 * XMARGIN_WALL - 1) /
  (XSIZE_SUBDOMAIN + 2 * XMARGIN_WALL),

  YTEXTURESIZE = 16 * ((_YTEXTURESIZE + 15) / 16),

  _ZTEXTURESIZE = ((ZSIZE_SUBDOMAIN + 2 * ZMARGIN_WALL) * XTEXTURESIZE +
		   XSIZE_SUBDOMAIN + 2 * XMARGIN_WALL - 1) /
  (XSIZE_SUBDOMAIN + 2 * XMARGIN_WALL),

  ZTEXTURESIZE = 16 * ((_ZTEXTURESIZE + 15) / 16),

};

namespace SolidWallsKernel {
  texture<float, 3, hipReadModeElementType> texSDF;

  texture<float4, 1, hipReadModeElementType> texWallParticles;
  texture<int, 1, hipReadModeElementType> texWallCellStart, texWallCellCount;

  __global__ void interactions_3tpp(const float2 *const pp, const int np,
				    const int nsolid, float *const acc,
				    const float seed);
  void setup() {
    texSDF.normalized = 0;
    texSDF.filterMode = hipFilterModePoint;
    texSDF.mipmapFilterMode = hipFilterModePoint;
    texSDF.addressMode[0] = hipAddressModeWrap;
    texSDF.addressMode[1] = hipAddressModeWrap;
    texSDF.addressMode[2] = hipAddressModeWrap;

    texWallParticles.channelDesc = hipCreateChannelDesc<float4>();
    texWallParticles.filterMode = hipFilterModePoint;
    texWallParticles.mipmapFilterMode = hipFilterModePoint;
    texWallParticles.normalized = 0;

    texWallCellStart.channelDesc = hipCreateChannelDesc<int>();
    texWallCellStart.filterMode = hipFilterModePoint;
    texWallCellStart.mipmapFilterMode = hipFilterModePoint;
    texWallCellStart.normalized = 0;

    texWallCellCount.channelDesc = hipCreateChannelDesc<int>();
    texWallCellCount.filterMode = hipFilterModePoint;
    texWallCellCount.mipmapFilterMode = hipFilterModePoint;
    texWallCellCount.normalized = 0;

    CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(interactions_3tpp), hipFuncCachePreferL1));
  }

  __device__ float sdf(float x, float y, float z) {
    int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};
    int MARGIN[3] = {XMARGIN_WALL, YMARGIN_WALL, ZMARGIN_WALL};
    int TEXSIZES[3] = {XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE};

    float tc[3], lmbd[3], r[3] = {x, y, z};
    for (int c = 0; c < 3; ++c) {
      float t =
	TEXSIZES[c] * (r[c] + L[c] / 2 + MARGIN[c]) / (L[c] + 2 * MARGIN[c]);

      lmbd[c] = t - (int)t;
      tc[c] = (int)t + 0.5;
    }
#define tex0(ix, iy, iz) (tex3D(texSDF, tc[0] + ix, tc[1] + iy, tc[2] + iz))
    float s000 = tex0(0, 0, 0), s001 = tex0(1, 0, 0), s010 = tex0(0, 1, 0);
    float s011 = tex0(1, 1, 0), s100 = tex0(0, 0, 1), s101 = tex0(1, 0, 1);
    float s110 = tex0(0, 1, 1), s111 = tex0(1, 1, 1);
#undef tex0

#define wavrg(A, B, p) A*(1-p) + p*B /* weighted average */
    float s00x = wavrg(s000, s001, lmbd[0]);
    float s01x = wavrg(s010, s011, lmbd[0]);
    float s10x = wavrg(s100, s101, lmbd[0]);
    float s11x = wavrg(s110, s111, lmbd[0]);

    float s0yx = wavrg(s00x, s01x, lmbd[1]);

    float s1yx = wavrg(s10x, s11x, lmbd[1]);
    float szyx = wavrg(s0yx, s1yx, lmbd[2]);
#undef wavrg
    return szyx;
  }

  __device__ float cheap_sdf(float x, float y, float z) // within the
							// rescaled
							// texel width
							// error
  {
    int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};
    int MARGIN[3] = {XMARGIN_WALL, YMARGIN_WALL, ZMARGIN_WALL};
    int TEXSIZES[3] = {XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE};

    float tc[3], r[3] = {x, y, z};;
    for (int c = 0; c < 3; ++c)
      tc[c] = 0.5001f + (int)(TEXSIZES[c] * (r[c] + L[c] / 2 + MARGIN[c]) /
			      (L[c] + 2 * MARGIN[c]));
#define tex0(ix, iy, iz) (tex3D(texSDF, tc[0] + ix, tc[1] + iy, tc[2] + iz))
    return tex0(0, 0, 0);
#undef  tex0
  }

  __device__ float3 ugrad_sdf(float x, float y, float z) {
    int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};
    int MARGIN[3] = {XMARGIN_WALL, YMARGIN_WALL, ZMARGIN_WALL};
    int TEXSIZES[3] = {XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE};

    float tc[3], fcts[3], r[3] = {x, y, z};
    for (int c = 0; c < 3; ++c)
      tc[c] = 0.5001f + (int)(TEXSIZES[c] * (r[c] + L[c] / 2 + MARGIN[c]) /
			      (L[c] + 2 * MARGIN[c]));
    for (int c = 0; c < 3; ++c) fcts[c] = TEXSIZES[c] / (2 * MARGIN[c] + L[c]);

#define tex0(ix, iy, iz) (tex3D(texSDF, tc[0] + ix, tc[1] + iy, tc[2] + iz))
    float myval = tex0(0, 0, 0);
    float gx = fcts[0] * (tex0(1, 0, 0) - myval);
    float gy = fcts[1] * (tex0(0, 1, 0) - myval);
    float gz = fcts[2] * (tex0(0, 0, 1) - myval);
#undef tex0

    return make_float3(gx, gy, gz);
  }

  __device__ float3 grad_sdf(float x, float y, float z) {
    int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};
    int MARGIN[3] = {XMARGIN_WALL, YMARGIN_WALL, ZMARGIN_WALL};
    int TEXSIZES[3] = {XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE};

    float tc[3], r[3] = {x, y, z};
    for (int c = 0; c < 3; ++c)
      tc[c] =
	TEXSIZES[c] * (r[c] + L[c] / 2 + MARGIN[c]) / (L[c] + 2 * MARGIN[c]);

    float gx, gy, gz;
#define tex0(ix, iy, iz) (tex3D(texSDF, tc[0] + ix, tc[1] + iy, tc[2] + iz))
    gx = tex0(1, 0, 0) - tex0(-1,  0,  0);
    gy = tex0(0, 1, 0) - tex0( 0, -1,  0);
    gz = tex0(0, 0, 1) - tex0( 0,  0, -1);
#undef tex0

    float ggmag = sqrt(gx*gx + gy*gy + gz*gz);

    if (ggmag > 1e-6) {
      gx /= ggmag; gy /= ggmag; gz /= ggmag;
    }
    return make_float3(gx, gy, gz);
  }

  __global__ void fill_keys(const Particle *const pp, const int n,
			    int *const key) {
    int pid = threadIdx.x + blockDim.x * blockIdx.x;

    if (pid >= n) return;

    Particle p = pp[pid];

    float sdf0 = sdf(p.x[0], p.x[1], p.x[2]);
    key[pid] = (int)(sdf0 >= 0) + (int)(sdf0 > 2);
  }

  __global__ void strip_solid4(Particle *const src, const int n, float4 *dst) {
    int pid = threadIdx.x + blockDim.x * blockIdx.x;
    if (pid >= n) return;
    Particle p = src[pid];
    dst[pid] = make_float4(p.x[0], p.x[1], p.x[2], 0);
  }

  __device__ void wall_vell(float x, float y, float z,
			    float *vxw, float *vyw, float *vzw) {
    *vxw = gamma_dot * z; *vyw = 0; *vzw = 0; /* velocity of the wall;
						 TODO: works only for
						 one processor */
  }

  __device__ void bounce_vel(float    x, float    y, float    z,
			     float* vxp, float* vyp, float* vzp) {
    float vx = *vxp,  vy = *vyp, vz = *vzp;

    float vxw, vyw, vzw; wall_vell(x, y, z, &vxw, &vyw, &vzw);

    vx -= vxw; vx = -vx; vx += vxw;
    vy -= vyw; vy = -vy; vy += vyw;
    vz -= vzw; vz = -vz; vz += vzw;

    last_bit_float::Preserver up1(*vxp);
    *vxp = vx; *vyp = vy; *vzp = vz;
  }

  __device__ void handle_collision(float currsdf,
				   float &x, float &y, float &z,
				   float &vx, float &vy, float &vz,
				   float dt) {
    float x0 = x - vx*dt, y0 = y - vy*dt, z0 = z - vz*dt;
    if (sdf(x0, y0, z0) >= 0) {
      // this is the worst case - it means that 0 position was bad already
      // we need to search and rescue the particle
      float3 gg = grad_sdf(x, y, z);
      float sdf0 = currsdf;
      x -= sdf0 * gg.x; y -= sdf0 * gg.y; z -= sdf0 * gg.z;
      for (int l = 8; l >= 1; --l) {
	if (sdf(x, y, z) < 0) {
	  bounce_vel(x, y, z, &vx, &vy, &vz); return;
	}
	float jump = 1.1f * sdf0 / (1 << l);
	x -= jump * gg.x; y -= jump * gg.y; z -= jump * gg.z;
      }
    }

    // newton raphson steps
    float subdt = dt;
    {
      float3 gg = ugrad_sdf(x, y, z);
      float DphiDt = max(1e-4f, gg.x * vx + gg.y * vy + gg.z * vz);
      subdt = min(dt, max(0.f, subdt - currsdf / DphiDt * 1.02f));
    }

    {
      float3 xstar = make_float3(x + subdt * vx, y + subdt * vy, z + subdt * vz);
      float3 gg = ugrad_sdf(xstar.x, xstar.y, xstar.z);
      float DphiDt = max(1e-4f, gg.x * vx + gg.y * vy + gg.z * vz);
      subdt = min(
		  dt, max(0.f, subdt - sdf(xstar.x, xstar.y, xstar.z) / DphiDt * 1.02f));
    }

    float lmbd = 2 * subdt - dt;
    x = x0 + lmbd * vx; y = y0 + lmbd * vy; z = z0 + lmbd * vz;
    bounce_vel(x, y, z, &vx, &vy, &vz);
    if (sdf(x, y, z) >= 0) {
      x = x0; y = y0; z = z0;
    }
  }

  __global__ void bounce(float2 *const pp, int nparticles, float dt) {
    int pid = threadIdx.x + blockDim.x * blockIdx.x;

    if (pid >= nparticles) return;

    float2 data0 = pp[pid * 3];
    float2 data1 = pp[pid * 3 + 1];
    if (pid < nparticles) {
      float mycheapsdf = cheap_sdf(data0.x, data0.y, data1.x);

      if (mycheapsdf >=
	  -1.7320f * ((float)XSIZE_WALLCELLS / (float)XTEXTURESIZE)) {
	float currsdf = sdf(data0.x, data0.y, data1.x);

	float2 data2 = pp[pid * 3 + 2];

	float3 v0 = make_float3(data1.y, data2.x, data2.y);

	if (currsdf >= 0) {
	  handle_collision(currsdf, data0.x, data0.y, data1.x, data1.y, data2.x,
			   data2.y, dt);

	  pp[3 * pid] = data0;
	  pp[3 * pid + 1] = data1;
	  pp[3 * pid + 2] = data2;
	}
      }
    }
  }

  __global__ void interactions_3tpp(const float2 *const pp, const int np,
				    const int nsolid, float *const acc,
				    const float seed) {
    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    int pid = gid / 3;
    int zplane = gid % 3;

    if (pid >= np) return;

    float2 dst0 = pp[3 * pid + 0];
    float2 dst1 = pp[3 * pid + 1];

    float interacting_threshold =
      -1 - 1.7320f * ((float)XSIZE_WALLCELLS / (float)XTEXTURESIZE);

    if (cheap_sdf(dst0.x, dst0.y, dst1.x) <= interacting_threshold) return;

    float2 dst2 = pp[3 * pid + 2];

    uint scan1, scan2, ncandidates, spidbase;
    int deltaspid1, deltaspid2;

    {
      int xbase = (int)(dst0.x - (-XSIZE_SUBDOMAIN / 2 - XMARGIN_WALL));
      int ybase = (int)(dst0.y - (-YSIZE_SUBDOMAIN / 2 - YMARGIN_WALL));
      int zbase = (int)(dst1.x - (-ZSIZE_SUBDOMAIN / 2 - ZMARGIN_WALL));

      enum {
	XCELLS = XSIZE_SUBDOMAIN + 2 * XMARGIN_WALL,
	YCELLS = YSIZE_SUBDOMAIN + 2 * YMARGIN_WALL,
	ZCELLS = ZSIZE_SUBDOMAIN + 2 * ZMARGIN_WALL,
	NCELLS = XCELLS * YCELLS * ZCELLS
      };

      int cid0 = xbase - 1 + XCELLS * (ybase - 1 + YCELLS * (zbase - 1 + zplane));

      spidbase = tex1Dfetch(texWallCellStart, cid0);
      int count0 = tex1Dfetch(texWallCellStart, cid0 + 3) - spidbase;

      int cid1 = cid0 + XCELLS;
      deltaspid1 = tex1Dfetch(texWallCellStart, cid1);
      int count1 = tex1Dfetch(texWallCellStart, cid1 + 3) - deltaspid1;

      int cid2 = cid0 + XCELLS * 2;
      deltaspid2 = tex1Dfetch(texWallCellStart, cid2);
      int count2 = cid2 + 3 == NCELLS
	? nsolid
	: tex1Dfetch(texWallCellStart, cid2 + 3) - deltaspid2;

      scan1 = count0;
      scan2 = count0 + count1;
      ncandidates = scan2 + count2;

      deltaspid1 -= scan1;
      deltaspid2 -= scan2;
    }

    float xforce = 0, yforce = 0, zforce = 0;

#define zig x
#define zag y

#define uno x
#define due y
#define tre z

#define mf3 make_float3
#define TYPE_WALL 3
    float  x = dst0.zig,  y = dst0.zag,  z = dst1.zig; /* bulk particle  */
    float vx = dst1.zag, vy = dst2.zig, vz = dst2.zag;

    for (int i = 0; i < ncandidates; ++i) {
      int m1 = (int)(i >= scan1);
      int m2 = (int)(i >= scan2);
      int spid = i + (m2 ? deltaspid2 : m1 ? deltaspid1 : spidbase);
      float4 stmp0 = tex1Dfetch(texWallParticles, spid);

      float  xw = stmp0.uno,  yw = stmp0.due,  zw = stmp0.tre; /* wall particle */
      float vxw, vyw, vzw; wall_vell(xw, yw, zw, &vxw, &vyw, &vzw);
      float rnd = Logistic::mean0var1(seed, pid, spid);

      // check for particle types and compute the DPD force
      int type_bulk = last_bit_float::get(vx);
      float3 strength = compute_dpd_force_traced(type_bulk      , TYPE_WALL,
						 mf3(x ,  y,  z), mf3( xw,  yw,  zw),
						 mf3(vx, vy, vz), mf3(vxw, vyw, vzw), rnd);
      xforce += strength.x; yforce += strength.y; zforce += strength.z;
    }
#undef zig
#undef zag

#undef uno
#undef due

#undef tre
#undef mf3
#undef TYPE_WALL

    atomicAdd(acc + 3 * pid + 0, xforce);
    atomicAdd(acc + 3 * pid + 1, yforce);
    atomicAdd(acc + 3 * pid + 2, zforce);
  }
}

template <int k> struct Bspline {
  template <int i> static float eval(float x) {
    return (x - i) / (k - 1) * Bspline<k - 1>::template eval<i>(x) +
      (i + k - x) / (k - 1) * Bspline<k - 1>::template eval<i + 1>(x);
  }
};

template <> struct Bspline<1> {
  template <int i> static float eval(float x) {
    return (float)(i) <= x && x < (float)(i + 1);
  }
};

struct FieldSampler {
  float *data, extent[3];
  int N[3];

  FieldSampler(const char *path, MPI_Comm comm) {
    static size_t CHUNKSIZE = 1 << 25;

    int rank;
    MPI_CHECK(MPI_Comm_rank(comm, &rank));

    if (rank == 0) {
      char header[2048];

      FILE *fh = fopen(path, "rb");

      fread(header, 1, sizeof(header), fh);

      printf("root parsing header\n");
      int retval = sscanf(header, "%f %f %f\n%d %d %d\n", extent + 0,
			  extent + 1, extent + 2, N + 0, N + 1, N + 2);

      if (retval != 6) {
	printf("ooops something went wrong in reading %s.\n", path);
	exit(EXIT_FAILURE);
      }

      printf("broadcasting N\n");
      MPI_CHECK(MPI_Bcast(N, 3, MPI_INT, 0, comm));
      MPI_CHECK(MPI_Bcast(extent, 3, MPI_FLOAT, 0, comm));

      int nvoxels = N[0] * N[1] * N[2];

      data = new float[nvoxels];

      if (data == NULL) {
	printf("ooops bad allocation %s.\n", path);
	exit(EXIT_FAILURE);
      }

      int header_size = 0;

      for (int i = 0; i < sizeof(header); ++i)
	if (header[i] == '\n') {
	  if (header_size > 0) {
	    header_size = i + 1;
	    break;
	  }

	  header_size = i + 1;
	}

      fseek(fh, header_size, SEEK_SET);
      fread(data, sizeof(float), nvoxels, fh);

      fclose(fh);
      for (size_t i = 0; i < nvoxels; i += CHUNKSIZE) {
	size_t s = (i + CHUNKSIZE <= nvoxels) ? CHUNKSIZE : (nvoxels - i);
	MPI_CHECK(MPI_Bcast(data + i, s, MPI_FLOAT, 0, comm));
      }

    } else {
      MPI_CHECK(MPI_Bcast(N, 3, MPI_INT, 0, comm));
      MPI_CHECK(MPI_Bcast(extent, 3, MPI_FLOAT, 0, comm));
      int nvoxels = N[0] * N[1] * N[2];

      data = new float[nvoxels];

      for (size_t i = 0; i < nvoxels; i += CHUNKSIZE) {
	size_t s = (i + CHUNKSIZE <= nvoxels) ? CHUNKSIZE : (nvoxels - i);
	MPI_CHECK(MPI_Bcast(data + i, s, MPI_FLOAT, 0, comm));
      }
    }
  }

  void sample(const float start[3], const float spacing[3], const int nsize[3],
	      float *const output, const float amplitude_rescaling) {
    Bspline<4> bsp;

    for (int iz = 0; iz < nsize[2]; ++iz)
      for (int iy = 0; iy < nsize[1]; ++iy)
	for (int ix = 0; ix < nsize[0]; ++ix) {
	  float x[3] = {start[0] + (ix + 0.5f) * spacing[0] - 0.5f,
			start[1] + (iy + 0.5f) * spacing[1] - 0.5f,
			start[2] + (iz + 0.5f) * spacing[2] - 0.5f};

	  int anchor[3];
	  for (int c = 0; c < 3; ++c) anchor[c] = (int)floor(x[c]);

	  float w[3][4];
	  for (int c = 0; c < 3; ++c)
	    for (int i = 0; i < 4; ++i)
	      w[c][i] = bsp.eval<0>(x[c] - (anchor[c] - 1 + i) + 2);

	  float tmp[4][4];
	  for (int sz = 0; sz < 4; ++sz)
	    for (int sy = 0; sy < 4; ++sy) {
	      float s = 0;

	      for (int sx = 0; sx < 4; ++sx) {
		int l[3] = {sx, sy, sz};

		int g[3];
		for (int c = 0; c < 3; ++c)
		  g[c] = (l[c] - 1 + anchor[c] + N[c]) % N[c];

		s += w[0][sx] * data[g[0] + N[0] * (g[1] + N[1] * g[2])];
	      }

	      tmp[sz][sy] = s;
	    }

	  float partial[4];
	  for (int sz = 0; sz < 4; ++sz) {
	    float s = 0;

	    for (int sy = 0; sy < 4; ++sy) s += w[1][sy] * tmp[sz][sy];

	    partial[sz] = s;
	  }

	  float val = 0;
	  for (int sz = 0; sz < 4; ++sz) val += w[2][sz] * partial[sz];

	  output[ix + nsize[0] * (iy + nsize[1] * iz)] =
	    val * amplitude_rescaling;
	}
  }

  ~FieldSampler() { delete[] data; }
};

ComputeWall::ComputeWall(MPI_Comm cartcomm, Particle *const p, const int n,
			 int &nsurvived, ExpectedMessageSizes &new_sizes)
  : cartcomm(cartcomm), arrSDF(NULL), solid4(NULL), solid_size(0),
    cells(XSIZE_SUBDOMAIN + 2 * XMARGIN_WALL,
	  YSIZE_SUBDOMAIN + 2 * YMARGIN_WALL,
	  ZSIZE_SUBDOMAIN + 2 * ZMARGIN_WALL) {
  MPI_CHECK(MPI_Comm_rank(cartcomm, &myrank));

  MPI_CHECK(MPI_Cart_get(cartcomm, 3, dims, periods, coords));

  float *field = new float[XTEXTURESIZE * YTEXTURESIZE * ZTEXTURESIZE];

  FieldSampler sampler("sdf.dat", cartcomm);

  int L[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};
  int MARGIN[3] = {XMARGIN_WALL, YMARGIN_WALL, ZMARGIN_WALL};
  int TEXTURESIZE[3] = {XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE};

  if (myrank == 0) printf("sampling the geometry file...\n");

  {
    float start[3], spacing[3];
    for (int c = 0; c < 3; ++c) {
      start[c] = sampler.N[c] * (coords[c] * L[c] - MARGIN[c]) /
	(float)(dims[c] * L[c]);
      spacing[c] = sampler.N[c] * (L[c] + 2 * MARGIN[c]) /
	(float)(dims[c] * L[c]) / (float)TEXTURESIZE[c];
    }

    float amplitude_rescaling = (XSIZE_SUBDOMAIN /*+ 2 * XMARGIN_WALL*/) /
      (sampler.extent[0] / dims[0]);

    sampler.sample(start, spacing, TEXTURESIZE, field, amplitude_rescaling);
  }

  if (myrank == 0) printf("redistancing the geometry field...\n");

  // extra redistancing because margin might exceed the domain
  {
    double dx = (XSIZE_SUBDOMAIN + 2 * XMARGIN_WALL) / (double)XTEXTURESIZE;
    double dy = (YSIZE_SUBDOMAIN + 2 * YMARGIN_WALL) / (double)YTEXTURESIZE;
    double dz = (ZSIZE_SUBDOMAIN + 2 * ZMARGIN_WALL) / (double)ZTEXTURESIZE;

    redistancing(field, XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE, dx, dy, dz,
		 XTEXTURESIZE * 2);
  }

  if (myrank == 0) printf("estimating geometry-based message sizes...\n");

  {
    for (int dz = -1; dz <= 1; ++dz)
      for (int dy = -1; dy <= 1; ++dy)
	for (int dx = -1; dx <= 1; ++dx) {
	  int d[3] = {dx, dy, dz};
	  int entry = (dx + 1) + 3 * ((dy + 1) + 3 * (dz + 1));

	  int local_start[3] = {d[0] + (d[0] == 1) * (XSIZE_SUBDOMAIN - 2),
				d[1] + (d[1] == 1) * (YSIZE_SUBDOMAIN - 2),
				d[2] + (d[2] == 1) * (ZSIZE_SUBDOMAIN - 2)};

	  int local_extent[3] = {1 * (d[0] != 0 ? 2 : XSIZE_SUBDOMAIN),
				 1 * (d[1] != 0 ? 2 : YSIZE_SUBDOMAIN),
				 1 * (d[2] != 0 ? 2 : ZSIZE_SUBDOMAIN)};

	  float start[3], spacing[3];
	  for (int c = 0; c < 3; ++c) {
	    start[c] = (coords[c] * L[c] + local_start[c]) /
	      (float)(dims[c] * L[c]) * sampler.N[c];
	    spacing[c] = sampler.N[c] / (float)(dims[c] * L[c]);
	  }

	  int nextent = local_extent[0] * local_extent[1] * local_extent[2];
	  float *data = new float[nextent];

	  sampler.sample(start, spacing, local_extent, data, 1);

	  int s = 0;
	  for (int i = 0; i < nextent; ++i) s += (data[i] < 0);

	  delete[] data;
	  double avgsize =
	    ceil(s * numberdensity /
		 (double)pow(2, abs(d[0]) + abs(d[1]) + abs(d[2])));

	  new_sizes.msgsizes[entry] = (int)avgsize;
	}
  }

  if (hdf5field_dumps) {
    if (myrank == 0) printf("H5 data dump of the geometry...\n");

    float *walldata =
      new float[XSIZE_SUBDOMAIN * YSIZE_SUBDOMAIN * ZSIZE_SUBDOMAIN];

    float start[3], spacing[3];
    for (int c = 0; c < 3; ++c) {
      start[c] = coords[c] * L[c] / (float)(dims[c] * L[c]) * sampler.N[c];
      spacing[c] = sampler.N[c] / (float)(dims[c] * L[c]);
    }

    int size[3] = {XSIZE_SUBDOMAIN, YSIZE_SUBDOMAIN, ZSIZE_SUBDOMAIN};

    float amplitude_rescaling = L[0] / (sampler.extent[0] / dims[0]);
    sampler.sample(start, spacing, size, walldata, amplitude_rescaling);

    H5FieldDump dump(cartcomm);
    dump.dump_scalarfield(cartcomm, walldata, "wall");

    delete[] walldata;
  }

  CUDA_CHECK(hipPeekAtLastError());

  hipChannelFormatDesc fmt = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipMalloc3DArray(
			       &arrSDF, &fmt,
			       make_hipExtent(XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE)));

  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr(
					  (void *)field, XTEXTURESIZE * sizeof(float), XTEXTURESIZE, YTEXTURESIZE);
  copyParams.dstArray = arrSDF;
  copyParams.extent = make_hipExtent(XTEXTURESIZE, YTEXTURESIZE, ZTEXTURESIZE);
  copyParams.kind = hipMemcpyHostToDevice;
  CUDA_CHECK(hipMemcpy3D(&copyParams));
  delete[] field;

  SolidWallsKernel::setup();

  CUDA_CHECK(hipBindTextureToArray(SolidWallsKernel::texSDF, arrSDF, fmt));

  if (myrank == 0) printf("carving out wall particles...\n");

  thrust::device_vector<int> keys(n);

  SolidWallsKernel::fill_keys<<<(n + 127) / 128, 128>>>(
							p, n, thrust::raw_pointer_cast(&keys[0]));
  CUDA_CHECK(hipPeekAtLastError());

  thrust::sort_by_key(keys.begin(), keys.end(),
		      thrust::device_ptr<Particle>(p));

  nsurvived = thrust::count(keys.begin(), keys.end(), 0);

  int nbelt = thrust::count(keys.begin() + nsurvived, keys.end(), 1);

  thrust::device_vector<Particle> solid_local(
					      thrust::device_ptr<Particle>(p + nsurvived),
					      thrust::device_ptr<Particle>(p + nsurvived + nbelt));

  if (hdf5part_dumps) {
    int n = solid_local.size();

    Particle *phost = new Particle[n];

    CUDA_CHECK(hipMemcpy(phost, thrust::raw_pointer_cast(&solid_local[0]),
			  sizeof(Particle) * n, hipMemcpyDeviceToHost));

    H5PartDump solid_dump("solid-walls.h5part", cartcomm, cartcomm);
    solid_dump.dump(phost, n);

    delete[] phost;
  }

  // can't use halo-exchanger class because of MARGIN
  // HaloExchanger halo(cartcomm, L, 666);
  // SimpleDeviceBuffer<Particle> solid_remote;
  // halo.exchange(thrust::raw_pointer_cast(&solid_local[0]),
  // solid_local.size(), solid_remote);

  if (myrank == 0)
    printf("fetching remote wall particles in my proximity...\n");

  SimpleDeviceBuffer<Particle> solid_remote;

  {
    thrust::host_vector<Particle> local = solid_local;

    int dstranks[26], remsizes[26], recv_tags[26];
    for (int i = 0; i < 26; ++i) {
      int d[3] = {(i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1};

      recv_tags[i] =
	(2 - d[0]) % 3 + 3 * ((2 - d[1]) % 3 + 3 * ((2 - d[2]) % 3));

      int coordsneighbor[3];
      for (int c = 0; c < 3; ++c) coordsneighbor[c] = coords[c] + d[c];

      MPI_CHECK(MPI_Cart_rank(cartcomm, coordsneighbor, dstranks + i));
    }

    // send local counts - receive remote counts
    {
      for (int i = 0; i < 26; ++i) remsizes[i] = -1;

      MPI_Request reqrecv[26];
      for (int i = 0; i < 26; ++i)
	MPI_CHECK(MPI_Irecv(remsizes + i, 1, MPI_INTEGER, dstranks[i],
			    123 + recv_tags[i], cartcomm, reqrecv + i));

      int localsize = local.size();

      MPI_Request reqsend[26];
      for (int i = 0; i < 26; ++i)
	MPI_CHECK(MPI_Isend(&localsize, 1, MPI_INTEGER, dstranks[i], 123 + i,
			    cartcomm, reqsend + i));

      MPI_Status statuses[26];
      MPI_CHECK(MPI_Waitall(26, reqrecv, statuses));
      MPI_CHECK(MPI_Waitall(26, reqsend, statuses));
    }

    std::vector<Particle> remote[26];

    // send local data - receive remote data
    {
      for (int i = 0; i < 26; ++i) remote[i].resize(remsizes[i]);

      MPI_Request reqrecv[26];
      for (int i = 0; i < 26; ++i)
	MPI_CHECK(MPI_Irecv(remote[i].data(), remote[i].size() * 6, MPI_FLOAT,
			    dstranks[i], 321 + recv_tags[i], cartcomm,
			    reqrecv + i));

      MPI_Request reqsend[26];
      for (int i = 0; i < 26; ++i)
	MPI_CHECK(MPI_Isend(local.data(), local.size() * 6, MPI_FLOAT,
			    dstranks[i], 321 + i, cartcomm, reqsend + i));

      MPI_Status statuses[26];
      MPI_CHECK(MPI_Waitall(26, reqrecv, statuses));
      MPI_CHECK(MPI_Waitall(26, reqsend, statuses));
    }

    // select particles within my region [-L / 2 - MARGIN, +L / 2 + MARGIN]
    std::vector<Particle> selected;
    for (int i = 0; i < 26; ++i) {
      int d[3] = {(i + 2) % 3 - 1, (i / 3 + 2) % 3 - 1, (i / 9 + 2) % 3 - 1};

      for (int j = 0; j < remote[i].size(); ++j) {
	Particle p = remote[i][j];

	for (int c = 0; c < 3; ++c) p.x[c] += d[c] * L[c];

	bool inside = true;

	for (int c = 0; c < 3; ++c)
	  inside &=
	    p.x[c] >= -L[c] / 2 - MARGIN[c] && p.x[c] < L[c] / 2 + MARGIN[c];

	if (inside) selected.push_back(p);
      }
    }

    solid_remote.resize(selected.size());
    CUDA_CHECK(hipMemcpy(solid_remote.data, selected.data(),
			  sizeof(Particle) * solid_remote.size,
			  hipMemcpyHostToDevice));
  }

  solid_size = solid_local.size() + solid_remote.size;

  Particle *solid;
  CUDA_CHECK(hipMalloc(&solid, sizeof(Particle) * solid_size));
  CUDA_CHECK(hipMemcpy(solid, thrust::raw_pointer_cast(&solid_local[0]),
			sizeof(Particle) * solid_local.size(),
			hipMemcpyDeviceToDevice));
  CUDA_CHECK(hipMemcpy(solid + solid_local.size(), solid_remote.data,
			sizeof(Particle) * solid_remote.size,
			hipMemcpyDeviceToDevice));

  if (solid_size > 0) cells.build(solid, solid_size, 0);

  CUDA_CHECK(hipMalloc(&solid4, sizeof(float4) * solid_size));

  if (myrank == 0) printf("consolidating wall particles...\n");

  if (solid_size > 0)
    SolidWallsKernel::strip_solid4<<<(solid_size + 127) / 128, 128>>>(
								      solid, solid_size, solid4);

  CUDA_CHECK(hipFree(solid));

  CUDA_CHECK(hipPeekAtLastError());

  frcs.resize(round(1.2 * n / 32.0));
  CUDA_CHECK(hipMemset(frcs.data, 0, frcs.size * sizeof(float3)));
  samples = 0;
}

void ComputeWall::bounce(Particle *const p, const int n, hipStream_t stream) {
  if (n > 0)
    SolidWallsKernel::bounce<<<(n + 127) / 128, 128, 0, stream>>>(
								  (float2 *)p, n, dt);

  samples++;
  CUDA_CHECK(hipPeekAtLastError());
}

void ComputeWall::interactions(const Particle *const p, const int n,
			       Acceleration *const acc,
			       hipStream_t stream) {
  // cellsstart and cellscount IGNORED for now

  if (n > 0 && solid_size > 0) {
    size_t textureoffset;
    CUDA_CHECK(hipBindTexture(&textureoffset,
			       &SolidWallsKernel::texWallParticles, solid4,
			       &SolidWallsKernel::texWallParticles.channelDesc,
			       sizeof(float4) * solid_size));

    CUDA_CHECK(hipBindTexture(&textureoffset,
			       &SolidWallsKernel::texWallCellStart, cells.start,
			       &SolidWallsKernel::texWallCellStart.channelDesc,
			       sizeof(int) * cells.ncells));

    CUDA_CHECK(hipBindTexture(&textureoffset,
			       &SolidWallsKernel::texWallCellCount, cells.count,
			       &SolidWallsKernel::texWallCellCount.channelDesc,
			       sizeof(int) * cells.ncells));

    SolidWallsKernel::
      interactions_3tpp<<<(3 * n + 127) / 128, 128, 0, stream>>>(
								 (float2 *)p, n, solid_size, (float *)acc, trunk.get_float());

    CUDA_CHECK(hipUnbindTexture(SolidWallsKernel::texWallParticles));
    CUDA_CHECK(hipUnbindTexture(SolidWallsKernel::texWallCellStart));
    CUDA_CHECK(hipUnbindTexture(SolidWallsKernel::texWallCellCount));
  }

  CUDA_CHECK(hipPeekAtLastError());
}

ComputeWall::~ComputeWall() {
  CUDA_CHECK(hipUnbindTexture(SolidWallsKernel::texSDF));
  CUDA_CHECK(hipFreeArray(arrSDF));
}
