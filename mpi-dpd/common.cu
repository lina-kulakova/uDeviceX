#include "hip/hip_runtime.h"
/*
 *  common.cu
 *  Part of uDeviceX/mpi-dpd/
 *
 *  Created and authored by Diego Rossinelli on 2015-01-30.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <mpi.h>
#include <sys/resource.h>
#include <cuda-dpd.h>
#include <cstdio>
#include ".conf.h" /* configuration file (copy from .conf.test.h) */
#include "common.h"

bool Particle::initialized = false;

MPI_Datatype Particle::mytype;

bool Acceleration::initialized = false;

MPI_Datatype Acceleration::mytype;

void CellLists::build(Particle * const p, const int n, hipStream_t stream, int * const order, const Particle * const src)
{
    if (n > 0)
      build_clists_vanilla((float * )p, n, 1, LX, LY, LZ, -LX/2, -LY/2, -LZ/2, order, start, count,  NULL, stream, (float *)src);
    else
    {
        CC(hipMemsetAsync(start, 0, sizeof(int) * ncells, stream));
        CC(hipMemsetAsync(count, 0, sizeof(int) * ncells, stream));
    }
}


void diagnostics(MPI_Comm comm, MPI_Comm cartcomm, Particle * particles, int n, float dt, int idstep, Acceleration * acc)
{
    double p[] = {0, 0, 0};
    for(int i = 0; i < n; ++i)
        for(int c = 0; c < 3; ++c)
            p[c] += particles[i].u[c];

    int rank;
    MPI_CHECK( MPI_Comm_rank(comm, &rank) );

    int dims[3], periods[3], coords[3];
    MPI_CHECK( MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    MPI_CHECK( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &p, rank == 0 ? &p : NULL, 3, MPI_DOUBLE, MPI_SUM, 0, comm) );

    double ke = 0;
    for(int i = 0; i < n; ++i)
        ke += pow(particles[i].u[0], 2) + pow(particles[i].u[1], 2) + pow(particles[i].u[2], 2);

    MPI_CHECK( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &ke, &ke, 1, MPI_DOUBLE, MPI_SUM, 0, comm) );
    MPI_CHECK( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &n, &n, 1, MPI_INT, MPI_SUM, 0, comm) );

    double kbt = 0.5 * ke / (n * 3. / 2);

    if (rank == 0)
    {
        static bool firsttime = true;
        FILE * f = fopen("diag.txt", firsttime ? "w" : "a");
        firsttime = false;

        if (idstep == 0)
            fprintf(f, "# TSTEP\tKBT\tPX\tPY\tPZ\n");

        printf("\x1b[91m timestep: %e\t%.10e\t%.10e\t%.10e\t%.10e\x1b[0m\n", idstep * dt, kbt, p[0], p[1], p[2]);
        fprintf(f, "%e\t%.10e\t%.10e\t%.10e\t%.10e\n", idstep * dt, kbt, p[0], p[1], p[2]);

        fclose(f);
    }
}

inline size_t hash_string(const char *buf)
{
    size_t result = 0;
    while( *buf != 0 ) {
        result = result * 31 + *buf++;
    }

    return result;
}


LocalComm::LocalComm()
{
    local_comm = MPI_COMM_NULL;
    local_rank = 0;
    local_nranks = 1;
}

void LocalComm::initialize(MPI_Comm _active_comm)
{
    active_comm = _active_comm;
    MPI_Comm_rank(active_comm, &rank);
    MPI_Comm_size(active_comm, &nranks);

    local_comm = active_comm;

    MPI_Get_processor_name(name, &len);
    size_t id = hash_string(name);

    MPI_Comm_split(active_comm, id, rank, &local_comm) ;

    MPI_Comm_rank(local_comm, &local_rank);
    MPI_Comm_size(local_comm, &local_nranks);
}

void LocalComm::barrier()
{
    if (!is_mps_enabled || local_nranks == 1) return;

    MPI_CHECK(MPI_Barrier(local_comm));
}
