#include "hip/hip_runtime.h"
#include <mpi.h>
#include <sys/resource.h>
#include <utility>
#include <cell-lists.h>
#include <cuda-dpd.h>
#include <cstdio>
#include ".conf.h" /* configuration file (copy from .conf.test.h) */
#include "common.h"
#include "common.tmp.h"

bool Particle::initialized = false;
MPI_Datatype Particle::mytype;

void CellLists::build(Particle * const p, const int n, int * const order, const Particle * const src)
{
    if (n > 0)
      build_clists_vanilla((float * )p, n, 1, LX, LY, LZ, -LX/2, -LY/2, -LZ/2, order, start, count,  NULL, (float *)src);
    else
    {
        CC(hipMemsetAsync(start, 0, sizeof(int) * ncells));
        CC(hipMemsetAsync(count, 0, sizeof(int) * ncells));
    }
}


void diagnostics(MPI_Comm comm, MPI_Comm cartcomm, Particle * particles, int n, float dt, int idstep) {
    double p[] = {0, 0, 0};
    for(int i = 0; i < n; ++i)
        for(int c = 0; c < 3; ++c)
            p[c] += particles[i].v[c];

    int rank;
    MC(MPI_Comm_rank(comm, &rank) );

    int dims[3], coords[3];
    MC(MPI_Cart_get(cartcomm, 3, dims, periods, coords) );

    MC(MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &p, rank == 0 ? &p : NULL, 3, MPI_DOUBLE, MPI_SUM, 0, comm) );

    double ke = 0;
    for(int i = 0; i < n; ++i)
        ke += pow(particles[i].v[0], 2) + pow(particles[i].v[1], 2) + pow(particles[i].v[2], 2);

    MC( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &ke, &ke, 1, MPI_DOUBLE, MPI_SUM, 0, comm) );
    MC( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &n, &n, 1, MPI_INT, MPI_SUM, 0, comm) );

    double kbt = 0.5 * ke / (n * 3. / 2);

    if (rank == 0)
    {
        static bool firsttime = true;
        FILE * f = fopen("diag.txt", firsttime ? "w" : "a");
        firsttime = false;

        if (idstep == 0)
            fprintf(f, "# TSTEP\tKBT\tPX\tPY\tPZ\n");

        printf("\x1b[91m timestep: %e\t%.10e\t%.10e\t%.10e\t%.10e\x1b[0m\n", idstep * dt, kbt, p[0], p[1], p[2]);
        fprintf(f, "%e\t%.10e\t%.10e\t%.10e\t%.10e\n", idstep * dt, kbt, p[0], p[1], p[2]);

        fclose(f);
    }
}
